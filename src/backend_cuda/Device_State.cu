
#include <hip/hip_runtime.h>
#ifdef BACKEND_CUDA

#include "Device_State.hpp"
#include "Host_State.hpp"
#include "backend_cuda/cuda_helper_functions.hpp"

namespace Spirit
{
namespace Device
{

void Device_State::allocate( Spirit::Host::Host_State * host_state )
{
    this->free();
    allocated = true;

    this->allocated     = true;
    this->nos           = host_state->nos;
    this->n_cells_total = host_state->n_cells_total;
    this->n_cell_atoms  = host_state->n_cell_atoms;
    this->n_cells[0]    = host_state->n_cells[0];
    this->n_cells[1]    = host_state->n_cells[1];
    this->n_cells[2]    = host_state->n_cells[2];

    this->n_ed = host_state->ed_stencils.size();
    this->n_k  = host_state->k_stencils.size();
    this->n_b  = host_state->b_stencils.size();

    this->timestep = host_state->timestep;
    CUDA_HELPER::malloc_n( this->gradient, host_state->nos );
    CUDA_HELPER::malloc_n( this->spins, host_state->nos );

    CUDA_HELPER::malloc_n( this->ed_stencils, host_state->ed_stencils.size() );
    CUDA_HELPER::malloc_n( this->k_stencils, host_state->k_stencils.size() );
    CUDA_HELPER::malloc_n( this->b_stencils, host_state->b_stencils.size() );
}

void Device_State::download( Spirit::Host::Host_State * host_state )
{
    CUDA_HELPER::copy_vector_D2H( host_state->spins, this->spins );
    CUDA_HELPER::copy_vector_D2H( host_state->gradient, this->gradient );
}

void Device_State::upload( Spirit::Host::Host_State * host_state )
{
    CUDA_HELPER::copy_vector_H2D( this->spins, host_state->spins );
    CUDA_HELPER::copy_vector_H2D( this->ed_stencils, host_state->ed_stencils );
    CUDA_HELPER::copy_vector_H2D( this->k_stencils, host_state->k_stencils );
    CUDA_HELPER::copy_vector_H2D( this->b_stencils, host_state->b_stencils );
}

void Device_State::free()
{
    if( !allocated )
        return;
    allocated = false;
    printf( "Freeing backend resources\n" );
    CUDA_HELPER::free( spins );
    CUDA_HELPER::free( gradient );
    CUDA_HELPER::free( ed_stencils );
}

} // namespace Device
} // namespace Spirit
#endif